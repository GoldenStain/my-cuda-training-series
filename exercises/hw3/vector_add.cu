
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <iomanip>

// error checking macro
#define cudaCheckErrors(msg)                                        \
    do {                                                            \
        hipError_t __err = hipGetLastError();                     \
        if (__err != hipSuccess) {                                 \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, \
                    hipGetErrorString(__err), __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n");             \
            exit(1);                                                \
        }                                                           \
    } while (0)

const int DSIZE = 32 * 1048576;
// vector add kernel: C = A + B
__global__ void vadd(const float *A, const float *B, float *C, int ds) {
    // 注意这里的步长是gridDim.x * blockDim.x ，这是我们第一次遇到gridDim.x
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < ds;
         idx += gridDim.x * blockDim.x) {  // a grid-stride loop
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    using namespace std::chrono;
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];  // allocate space for vectors in host memory
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];
    for (int i = 0; i < DSIZE; i++) {  // initialize vectors in host memory
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
        h_C[i] = 0;
    }
    hipMalloc(&d_A,
               DSIZE * sizeof(float));  // allocate device space for vector A
    hipMalloc(&d_B,
               DSIZE * sizeof(float));  // allocate device space for vector B
    hipMalloc(&d_C,
               DSIZE * sizeof(float));  // allocate device space for vector C
    cudaCheckErrors("hipMalloc failure");  // error checking
    // copy vector A to device:
    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    // copy vector B to device:
    hipMemcpy(d_B, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
    // cuda processing sequence step 1 is complete
    int threads = 512;  // modify this line for experimentation
    int blocks = 16;     // modify this line for experimentation
    auto start = high_resolution_clock::now();
    vadd<<<blocks, threads>>>(d_A, d_B, d_C, DSIZE);
    auto end = high_resolution_clock::now();
    duration<double, std::milli> elapsed = end - start;
    std::cout << "It took " << std::fixed << std::setprecision(3) << elapsed.count() << "milliseconds" << std::endl;
    cudaCheckErrors("kernel launch failure");
    // cuda processing sequence step 2 is complete
    //  copy vector C from device to host:
    hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    // cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    printf("A[0] = %f\n", h_A[0]);
    printf("B[0] = %f\n", h_B[0]);
    printf("C[0] = %f\n", h_C[0]);
    return 0;
}
