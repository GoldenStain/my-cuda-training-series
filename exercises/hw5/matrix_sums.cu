
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024
// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds){
    int bidx = blockIdx.x, base = bidx * ds; // 表示行号
    if (bidx < ds) {
        __shared__ float sdata[block_size];
        // 通过stride把整行都放进来
        int tidx = threadIdx.x, now_idx = tidx;
        sdata[tidx] = 0.0f;
        while(now_idx < ds) {
            sdata[tidx] += A[base+now_idx];
            now_idx += blockDim.x;
        }
        for (unsigned int offset = blockDim.x/2; offset; offset>>=1) {
            __syncthreads();
            if (tidx < offset) // 只有前一半需要累加
                sdata[tidx] += sdata[tidx + offset];
        }
        if (tidx == 0)
            sums[bidx] = sdata[0];
    }
}
// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx+ds*i];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}
bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}
int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  //cuda processing sequence step 1 is complete
  row_sums<<<DSIZE, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums correct!\n");
  hipMemset(d_sums, 0, DSIZE*sizeof(float));
  column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("column sums correct!\n");
  return 0;
}
  
