
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
    hello<<<20, 10>>>();
    hipDeviceSynchronize();
    printf("end of execution.\n");
}
