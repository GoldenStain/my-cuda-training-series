
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

using namespace std;

#define N 4096
#define RADIUS 3
#define BLOCK_SIZE 16

__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE+2*RADIUS];
    // 因为我们把地址传进来的时候已经加了RADIUS，这里gindex就不用加了。
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
      result += temp[lindex + offset];

    // Store the result
    // 因为我们的d_in , d_out只偏移了RADIUS，所以最后RADIUS个取不到，就不会做多余操作。
    out[gindex] = result;
}

void fill_ints(int *x, int n) {
  fill_n(x, n, 1);
}

int main(void) {
  int *in, *out; // host copies of a, b, c
  int *d_in, *d_out; // device copies of a, b, c

  // Alloc space for host copies and setup values
  int size = (N + 2*RADIUS) * sizeof(int);
  in = (int *)malloc(size); fill_ints(in, N + 2*RADIUS);
  out = (int *)malloc(size); fill_ints(out, N + 2*RADIUS);

  // Alloc space for device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

  // Launch stencil_1d() kernel on GPU
  // 注意这里也要加上偏移量RADIUS  不然两头就会偏大
  stencil_1d<<<N/BLOCK_SIZE,BLOCK_SIZE>>>(d_in+RADIUS, d_out+RADIUS);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // Error Checking
  for (int i = 0; i < N + 2*RADIUS; i++) {
    if (i<RADIUS || i>=N+RADIUS){
      if (out[i] != 1)
    	printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1);
    } else {
      if (out[i] != 1 + 2*RADIUS)
    	printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1 + 2*RADIUS);
    }
  }

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  printf("Success!\n");
  return 0;
}
